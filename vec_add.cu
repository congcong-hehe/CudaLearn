#include "hip/hip_runtime.h"
// 2022/1/13

#include "test.h"

#define N 100
#define M 10

__global__ void kernel1(int* a, int* b, int* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= N) return;
	c[index] = a[index] + b[index];
}

void vecAdd()
{
	int a[N];
	int b[N];
	int c[N];

	float elapsed_time = 0;
	hipEvent_t start, stop;

	for (int i = 0; i < N; ++i)
	{
		a[i] = b[i] = i;
	}

	int* d_a, * d_b, * d_c;

	hipMalloc((void**)&d_a, sizeof(int) * N);
	hipMalloc((void**)&d_b, sizeof(int) * N);
	hipMalloc((void**)&d_c, sizeof(int) * N);

	hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);

	START_GPU
	kernel1 << <(N + M -1) / M, M >> > (d_a, d_b, d_c);
	hipError_t err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	STOP_GPU

	printf("%fms\n", elapsed_time);

	hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i = N-1; i > N - 20; --i)
	{
		printf("%d\n", c[i]);
	}
}

